#include "hip/hip_runtime.h"
#include "a3.hpp"
#include <vector>
#include <cmath>
#include <functional>
#include <hip/hip_runtime.h>
#include <algorithm>
#include <iostream>

__global__ void computeKDE(int n, float h, const float* x, float* y) {
    extern __shared__ float sharedX[]; // Shared memory for 'x' values
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int tx = threadIdx.x;

    if (i < n) {
        // Load 'x' values into shared memory
        sharedX[tx] = x[i];
        __syncthreads(); // Synchronize to make sure all 'x' are loaded

        float sum = 0.0;
        float inv_h = 1.0 / h;
        for (int j = 0; j < n; j++) {
            float u = (sharedX[tx] - x[j]) * inv_h;
            sum += exp(-0.5 * u * u) / (sqrtf(2 * M_PI));
        }
        y[i] = sum / (n * h);
    }
}


void gaussian_kde(int n, float h, const std::vector<float>& x, std::vector<float>& y) {
    float *d_x, *d_y;

    hipMalloc(&d_x, n * sizeof(float));
    hipMalloc(&d_y, n * sizeof(float));

    hipMemcpy(d_x, x.data(), n * sizeof(float), hipMemcpyHostToDevice);

    // int threadsPerBlock = 256;
    // int blocks = (n + threadsPerBlock - 1) / threadsPerBlock;

    int threadsPerBlock = 256;
    int blocks = (n + threadsPerBlock - 1) / threadsPerBlock;
    int sharedMemSize = threadsPerBlock * sizeof(float);

    computeKDE<<<blocks, threadsPerBlock, sharedMemSize>>>(n, h, d_x, d_y);

    // computeKDE<<<blocks, threadsPerBlock>>>(n, h, d_x, d_y);
    // hipDeviceSynchronize();
    hipDeviceSynchronize();
hipError_t error = hipGetLastError();
if (error != hipSuccess) {
    // std::cerr << "CUDA Error: " << hipGetErrorString(error) << std::endl;
    // Handle error or exit
    fprintf(stderr, "CUDA Error: %s\n", hipGetErrorString(error));
}

    hipMemcpy(y.data(), d_y, n * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_x);
    hipFree(d_y);
}
